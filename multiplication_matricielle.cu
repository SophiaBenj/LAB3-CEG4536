#include "hip/hip_runtime.h"
%%writefile multiplication_matricielle.cu 

// étape 1

#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void matrixMul(const float *A, const float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    if (row < n && col < n) {
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void initializeMatrix(float *mat, int n, float value) {
    for (int i = 0; i < n * n; ++i) {
        mat[i] = value;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C;  // Matrices sur l'hôte
    float *d_A, *d_B, *d_C;  // Matrices sur le GPU

    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    initializeMatrix(h_A, N, 1.0f);
    initializeMatrix(h_B, N, 2.0f);

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    matrixMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Résultat (partie de la matrice C) :" << std::endl;
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}